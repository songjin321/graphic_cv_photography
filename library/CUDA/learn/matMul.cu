
#include <hip/hip_runtime.h>
#include <stdio.h>
// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
   } Matrix;
   // Thread block size
   #define BLOCK_SIZE 16
   // Forward declaration of the matrix multiplication kernel
   __global__ void MatMulKernel(const Matrix, const Matrix, Matrix);
   // Matrix multiplication - Host code
   // Matrix dimensions are assumed to be multiples of BLOCK_SIZE
   void MatMul(const Matrix A, const Matrix B, Matrix C)
   {
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
    hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);
    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);
    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    printf("Hello2");
    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
    hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
   }
   // Matrix multiplication kernel called by MatMul()
   __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
   {
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
    
   }

int main()
{
    float A_data[4] = {1,2,3,4};
    float B_data[4] = {5,6,7,8};
    Matrix A,B;
    A.height = 2;
    A.width = 2;
    A.elements = A_data;
    B.height = 2;
    B.width = 2;
    B.elements = B_data;
    Matrix C;
    C.height = 2;
    C.width = 2;
    MatMul(A,B,C);
    printf("%f %f %f %f", C.elements[0], C.elements[1], C.elements[2], C.elements[3]);
}